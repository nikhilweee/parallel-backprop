#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <random>
#include <string>

#include "matrix.h"

#define idx(i, j, N) ((i) * (N)) + (j)

void print_array(float* array, int size) {
  printf("array [%d]: ", size);
  for (int i = 0; i < size; i++) {
    printf("%.05f ", array[i]);
  }
  cout << endl;
}

////////////////////////////////////////////////////////////////////////////////

Matrix::Matrix(){};
Matrix::Matrix(vector2d& input) : data(input){};
Matrix::Matrix(int x, int y) {
  for (int i = 0; i < x; i++) {
    vector1d row;
    for (int j = 0; j < y; j++) {
      row.push_back(1.0);
    }
    this->data.push_back(row);
  }
}

////////////////////////////////////////////////////////////////////////////////

void Matrix::to_array() {
  int x = this->size()[0];
  int y = this->size()[1];
  int N = x * y;

  float* array = new float[N];
  for (int i = 0; (i < x); i++) {
    for (int j = 0; (j < y); j++) {
      array[idx(i, j, y)] = this->data[i][j];
    }
  }
  this->array = array;
}

void Matrix::from_array() {
  int x = this->size()[0];
  int y = this->size()[1];

  // TODO: check if size of array matches
  for (int i = 0; (i < x); i++) {
    for (int j = 0; (j < y); j++) {
      this->data[i][j] = this->array[idx(i, j, y)];
    }
  }
}

void Matrix::init_grad() {
  this->requires_grad = true;
  int x = this->size()[0];
  int y = this->size()[1];
  Matrix* grad = new Matrix(x, y);
  this->grad = grad;
}

////////////////////////////////////////////////////////////////////////////////

void Matrix::uniform(float a, float b) {
  int x = this->size()[0];
  int y = this->size()[1];
  // random_device rand_dev;
  // default_random_engine generator(rand_dev());
  default_random_engine generator;
  uniform_real_distribution<float> uniform(a, b);
  for (int i = 0; i < x; i++) {
    vector<float> row;
    for (int j = 0; j < y; j++) {
      this->data[i][j] = uniform(generator);
    }
  }
}

void Matrix::ones() {
  int x = this->size()[0];
  int y = this->size()[1];
  Matrix result(x, y);
  for (int i = 0; i < x; i++) {
    for (int j = 0; j < y; j++) {
      this->data[i][j] = 1.0;
    }
  }
};

Matrix Matrix::tanh() {
  int x = this->size()[0];
  int y = this->size()[1];
  Matrix result(x, y);
  for (int i = 0; i < x; i++) {
    for (int j = 0; j < y; j++) {
      result.data[i][j] = std::tanh(this->data[i][j]);
    }
  }
  return result;
};

Matrix Matrix::square() {
  int x = this->size()[0];
  int y = this->size()[1];
  Matrix result(x, y);

  if (this->requires_grad) {
    result.init_grad();
  }

  for (int i = 0; i < x; i++) {
    for (int j = 0; j < y; j++) {
      result.data[i][j] = this->data[i][j] * this->data[i][j];
    }
  }
  return result;
};

////////////////////////////////////////////////////////////////////////////////

__global__ void mul_kernel(float* result, float* self, float other, int N) {
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < N; i += stride) {
    result[i] = self[i] * other;
  }
}

Matrix Matrix::mul(float other) {
  int x = this->size()[0];
  int y = this->size()[1];
  int N = x * y;

  Matrix result(x, y);

  if (this->requires_grad) {
    result.init_grad();
  }

  int size = N * sizeof(float);

  float* self_cuda;
  hipMalloc((void**)&self_cuda, size);
  this->to_array();
  hipMemcpy(self_cuda, this->array, size, hipMemcpyHostToDevice);

  float* result_cuda;
  hipMalloc((void**)&result_cuda, size);
  result.to_array();
  hipMemcpy(result_cuda, result.array, size, hipMemcpyHostToDevice);

  int num_blocks = ceil((float)N / 512);
  mul_kernel<<<num_blocks, 512>>>(result_cuda, self_cuda, other, N);

  hipMemcpy(result.array, result_cuda, size, hipMemcpyDeviceToHost);

  hipFree(self_cuda);
  hipFree(result_cuda);

  result.from_array();

  return result;
};

////////////////////////////////////////////////////////////////////////////////

__global__ void mulip_kernel(float* self, float* other, int N) {
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < N; i += stride) {
    self[i] = self[i] * other[i];
  }
}

void Matrix::mulip(Matrix* other) {
  int x = this->size()[0];
  int y = this->size()[1];
  int N = x * y;

  if (this->requires_grad || other->requires_grad) {
    if (!this->requires_grad) {
      this->init_grad();
    }
    if (!other->requires_grad) {
      other->init_grad();
    }
  }

  int size = N * sizeof(float);

  float* self_cuda;
  hipMalloc((void**)&self_cuda, size);
  this->to_array();
  hipMemcpy(self_cuda, this->array, size, hipMemcpyHostToDevice);

  float* other_cuda;
  hipMalloc((void**)&other_cuda, size);
  other->to_array();
  hipMemcpy(other_cuda, other->array, size, hipMemcpyHostToDevice);

  mulip_kernel<<<1, 512>>>(self_cuda, other_cuda, N);

  hipMemcpy(this->array, self_cuda, size, hipMemcpyDeviceToHost);

  hipFree(self_cuda);
  hipFree(other_cuda);

  this->from_array();
};

////////////////////////////////////////////////////////////////////////////////

__global__ void transpose_kernel(float* result, float* self, int x, int y, int N) {
  int row = threadIdx.x + blockDim.x * blockIdx.x;
  int col = threadIdx.y + blockDim.y * blockIdx.y;

  if (idx(row, col, x) < N && idx(col, row, y) < N) {
    result[idx(row, col, x)] = self[idx(col, row, y)];
  }
}

Matrix Matrix::transpose() {
  int x = this->size()[0];
  int y = this->size()[1];
  int N = x * y;

  Matrix result = Matrix(y, x);

  int size = N * sizeof(float);

  float* self_cuda;
  hipMalloc((void**)&self_cuda, size);
  this->to_array();
  hipMemcpy(self_cuda, this->array, size, hipMemcpyHostToDevice);

  float* result_cuda;
  hipMalloc((void**)&result_cuda, size);
  result.to_array();
  hipMemcpy(result_cuda, result.array, size, hipMemcpyHostToDevice);

  dim3 num_threads(32, 32);
  dim3 num_blocks(1, 1);
  num_blocks.x = ceil((float)x / 512);
  num_blocks.y = ceil((float)y / 512);
  transpose_kernel<<<num_blocks, num_threads>>>(result_cuda, self_cuda, x, y, N);

  hipMemcpy(result.array, result_cuda, size, hipMemcpyDeviceToHost);

  hipFree(self_cuda);
  hipFree(result_cuda);

  result.from_array();

  return result;
};

////////////////////////////////////////////////////////////////////////////////

float Matrix::sum() {
  int x = this->size()[0];
  int y = this->size()[1];
  float sum = 0;
  for (int i = 0; i < x; i++) {
    for (int j = 0; j < y; j++) {
      sum += data[i][j];
    }
  }
  return sum;
};

Matrix Matrix::cols(int a, int b) {
  int x = this->size()[0];
  Matrix result = Matrix(x, b - a);
  for (int i = 0; i < x; i++) {
    for (int j = a; j < b; j++) {
      result.data[i][j - a] = this->data[i][j];
    }
  }
  return result;
};

////////////////////////////////////////////////////////////////////////////////
// Core operations
////////////////////////////////////////////////////////////////////////////////

__global__ void matmul_kernel(float* result, float* self, float* other, int x,
                              int y, int z) {
  int row = threadIdx.x + blockDim.x * blockIdx.x;
  int col = threadIdx.y + blockDim.y * blockIdx.y;
  float sum = 0.0;
  if (row < x && col < y) {
    for (int i = 0; i < z; i++) {
      sum += self[idx(row, i, z)] * other[idx(i, col, y)];
    }
    result[idx(row, col, y)] = sum;
  }
}

Matrix Matrix::matmul(Matrix& other) {
  if (size()[1] != other.size()[0]) {
    cout << "Sizes of matrices should be compatible. Got " << size_str()
         << " and " << other.size_str() << endl;
    exit(EXIT_FAILURE);
  }
  int x = this->size()[0];
  int y = other.size()[1];
  int z = this->size()[1];
  int N = x * y;

  Matrix result = Matrix(x, y);

  if (this->requires_grad || other.requires_grad) {
    if (!this->requires_grad) {
      this->init_grad();
    }
    if (!other.requires_grad) {
      other.init_grad();
    }
    result.init_grad();
  }

  int size_self = x * z * sizeof(float);
  int size_other = z * y * sizeof(float);
  int size_result = N * sizeof(float);

  float* self_cuda;
  hipMalloc((void**)&self_cuda, size_self);
  this->to_array();
  hipMemcpy(self_cuda, this->array, size_self, hipMemcpyHostToDevice);

  float* other_cuda;
  hipMalloc((void**)&other_cuda, size_other);
  other.to_array();
  hipMemcpy(other_cuda, other.array, size_other, hipMemcpyHostToDevice);

  float* result_cuda;
  hipMalloc((void**)&result_cuda, size_result);
  result.to_array();
  hipMemcpy(result_cuda, result.array, size_result, hipMemcpyHostToDevice);

  dim3 num_threads(32, 32);
  dim3 num_blocks(1, 1);
  num_blocks.x = ceil((float)x / 512);
  num_blocks.y = ceil((float)y / 512);
  matmul_kernel<<<num_blocks, num_threads>>>(result_cuda, self_cuda, other_cuda,
                                             x, y, z);

  hipMemcpy(result.array, result_cuda, size_result, hipMemcpyDeviceToHost);

  hipFree(self_cuda);
  hipFree(other_cuda);
  hipFree(result_cuda);

  result.from_array();
  return result;
};

////////////////////////////////////////////////////////////////////////////////

__global__ void add_kernel(float* result, float* self, float* other, int N) {
  int index = threadIdx.x + blockDim.x * blockIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < N; i += stride) {
    result[i] = self[i] + other[i];
  }
}

Matrix Matrix::add(Matrix other) {
  if (this->size()[1] != other.size()[1]) {
    cout << "Sizes of matrices should be compatible. Got " << size_str()
         << " and " << other.size_str() << endl;
    exit(EXIT_FAILURE);
  }

  int x = this->size()[0];
  int y = this->size()[1];
  int N = x * y;

  if (other.size()[0] == 1) {
    Matrix interm = Matrix(x, y);
    for (int i = 0; i < x; i++) {
      for (int j = 0; j < y; j++) {
        interm.data[i][j] = other.data[0][j];
      }
    }
    other = interm;
  }

  if (this->size()[0] != other.size()[0]) {
    cout << "Sizes of matrices should be compatible. Got " << size_str()
         << " and " << other.size_str() << endl;
    exit(EXIT_FAILURE);
  }

  Matrix result = Matrix(x, y);

  if (this->requires_grad || other.requires_grad) {
    if (!this->requires_grad) {
      this->init_grad();
    }
    if (!other.requires_grad) {
      other.init_grad();
    }
    result.init_grad();
  }

  int size = N * sizeof(float);

  float* self_cuda;
  hipMalloc((void**)&self_cuda, size);
  this->to_array();
  hipMemcpy(self_cuda, this->array, size, hipMemcpyHostToDevice);

  float* other_cuda;
  hipMalloc((void**)&other_cuda, size);
  other.to_array();
  hipMemcpy(other_cuda, other.array, size, hipMemcpyHostToDevice);

  float* result_cuda;
  hipMalloc((void**)&result_cuda, size);
  result.to_array();
  hipMemcpy(result_cuda, result.array, size, hipMemcpyHostToDevice);

  int num_blocks = ceil((float)N / 512);
  add_kernel<<<num_blocks, 512>>>(result_cuda, self_cuda, other_cuda, N);

  hipMemcpy(result.array, result_cuda, size, hipMemcpyDeviceToHost);

  hipFree(self_cuda);
  hipFree(other_cuda);
  hipFree(result_cuda);

  result.from_array();

  return result;
};

// Helper functions

void Matrix::print_data() {
  for (vector1d row : data) {
    for (float col : row) {
      printf("%10.4f ", col);
    }
    cout << endl;
  }
};

vector<int> Matrix::size() {
  vector<int> size;
  size.push_back(data.size());
  size.push_back(data[0].size());
  return size;
};

string Matrix::size_str() {
  vector<int> size_vec = size();
  string size_str;
  size_str = "(";
  for (size_t i = 0; i < size_vec.size(); i++) {
    size_str += to_string(size_vec[i]);
    if (i < (size_vec.size() - 1)) {
      size_str += ", ";
    }
  }
  size_str += ")";
  return size_str;
};

void Matrix::print_size() {
  // useless comment
  cout << size_str() << endl;
};